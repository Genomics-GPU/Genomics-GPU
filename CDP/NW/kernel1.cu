#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

#define BLOCK_DIM 32

__global__ void nw_gpu1_kernel (unsigned char * reference_d, unsigned char* query_d, int* matrix_d, unsigned int N, unsigned int round) {
	
	__shared__ unsigned int q_offset;
	__shared__ unsigned int r_offset;
	__shared__ unsigned int loop_limit;

	if(threadIdx.x == 0){
		//Check if it is round 1 or 2 in overall matrix of blocks
		if (round == 1){
			q_offset = BLOCK_DIM*blockIdx.x;
			r_offset = BLOCK_DIM*(gridDim.x - 1 - blockIdx.x);
		}
		else if (round == 2){
			q_offset = BLOCK_DIM*((N + BLOCK_DIM - 1)/BLOCK_DIM - gridDim.x + blockIdx.x );
			r_offset = BLOCK_DIM*((N + BLOCK_DIM - 1)/BLOCK_DIM - blockIdx.x - 1);	
		}
		//Loop limit is used as a boundary check
		//If the block is not complete and some elements are out of bounds, we can loop fewer times
		loop_limit = (((N-q_offset) > BLOCK_DIM && (N-r_offset) > BLOCK_DIM) || N%BLOCK_DIM == 0)? 2*BLOCK_DIM : ((N-q_offset) < BLOCK_DIM && (N-r_offset) < BLOCK_DIM)? 2*(N % BLOCK_DIM)  : BLOCK_DIM + N % BLOCK_DIM;
	}
	__syncthreads();

	for (int i = 1; i < loop_limit; i++){
		//Check if it is round 1 or 2 within the block
		int idx = (i < BLOCK_DIM + 1)? i : 2*BLOCK_DIM - i;
		int q = 0;
		int r = 0;
		if (i < BLOCK_DIM + 1) {
			//This is round 1;
			q = threadIdx.x + q_offset;
			r = idx - threadIdx.x - 1 + r_offset;
		}
		else {
			//This is round 2
			q = BLOCK_DIM - idx + threadIdx.x + q_offset;
			r = BLOCK_DIM - threadIdx.x - 1 + r_offset;
		}

		if(threadIdx.x < idx && q < N && r < N) {
                	int top     = (q == 0)?((r + 1)*DELETION):(matrix_d[(q - 1)*N + r]);
                	int left    = (r == 0)?((q + 1)*INSERTION):(matrix_d[q*N + (r - 1)]);
                	int topleft = (q == 0)?(r*DELETION):((r == 0)?(q*INSERTION):(matrix_d[(q - 1)*N + (r - 1)]));
                	// Find scores based on neighbors
                	int insertion = top + INSERTION;
                	int deletion  = left + DELETION;
                	int match     = topleft + ((query_d[q] == reference_d[r])?MATCH:MISMATCH);
                	// Select best score
                	int max = (insertion > deletion)?insertion:deletion;
                	max = (match > max)?match:max;
                	matrix_d[q*N + r] = max;
        	}
		__syncthreads();
	}
	
}


__global__ void nw_gpu1(unsigned char* reference_d, unsigned char* query_d, int* matrix_d, unsigned int N) {

	//Each tile is of dimension BLOCK_DIM*BLOCK_DIM
	//Max number of threads simultaneoulsy active in a tile is BLOCK_DIM
	//So number of threads per block is BLOCK_DIM	
	int numThreadsPerBlock = BLOCK_DIM;
	
	for (unsigned int i = 1; i < (N + BLOCK_DIM - 1)/BLOCK_DIM + 1; i++) {
		//Number of blocks (i.e. of tiles)  is equal to the iteration number
                int numBlocks = i;
                nw_gpu1_kernel <<< numBlocks, numThreadsPerBlock >>> (reference_d, query_d, matrix_d, N, 1);
             //   hipDeviceSynchronize();
        }
        for (int i = (N + BLOCK_DIM - 1)/BLOCK_DIM -1; i>0; i--){
                int numBlocks = i;
                nw_gpu1_kernel <<< numBlocks, numThreadsPerBlock >>> (reference_d, query_d, matrix_d, N, 2);
             //   hipDeviceSynchronize();
        }
}
