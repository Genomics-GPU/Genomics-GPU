	#include <iostream>
	#include<limits>
	#include <stdlib.h>
	#include <stdio.h>
	#include <string.h>
	#include <time.h>
	#include <hip/hip_runtime.h>
	#include <stdint.h>
	#include <math.h>
	#include <unistd.h>
	#include <omp.h>	
	 #include <algorithm>


	using namespace std;

	// 8 byte.   how to be 128byte?
	struct NUM_ADD
	{
		int read_number;
		int haplotype_number;	
	//	short2 read_haplotype;
		int  Read_array;
		int  read_large_length;
	};

	double diff(timespec start, timespec end)
	{
	  double a=0;
	 if((end.tv_nsec-start.tv_nsec)<0)
	{
	a=end.tv_sec-start.tv_sec-1;
	a+=(1000000000+end.tv_nsec-start.tv_nsec)/1000000000.0;
	}
	else
	{
	a=end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

	}
	return a;
	}

	__constant__ float  constant[10];
	__constant__ int  constant_int[10];
	

	__global__ void  pairHMM( int size, char * data,  NUM_ADD * num_add, float * result,float * MG,float * DG, float * IG ) // what is the maximum number of parameters?
	{
	//MG, DG and IG are global memory to store indermediate result?
	//each thread finish one computation		
	int offset=blockIdx.x*blockDim.x+threadIdx.x;
	MG=MG+offset;
	IG=IG+offset;
	DG=DG+offset;
	//if(threadIdx.x==0)
	//printf("%d %d %d %d %d\n", constant_int[0],constant_int[1], constant_int[2],constant_int[3], constant_int[4]);
	while(offset<size)
	 {	
		__shared__ float parameter1[256];
		__shared__ float parameter2[256];
		__shared__ float parameter3[256];
		__shared__ float parameter4[256];
	
		//NUM_ADD number_address;
		//number_address=num_add[offset];//get from global memory
		int read_large_length=num_add[offset].read_large_length;
		char4 * read_base_array=(char4 *)(data+num_add[offset].Read_array); // to caculate the address of read_base_array. 
		int  read_number=num_add[offset].read_number;
		int  haplotype_number=num_add[offset].haplotype_number;
		float  *parameter1_array=(float *) (read_base_array+(read_large_length+3)/4*32);
		read_large_length=read_large_length*32;
		float  *parameter2_array=(float *) (parameter1_array+read_large_length);
		float  *parameter3_array=(float *) (parameter1_array+read_large_length*2);
		float  *parameter4_array=(float *) (parameter1_array+read_large_length*3);
		char4 * haplotype_base_array=(char4 * )(parameter1_array+read_large_length*4);    	
		//haplotype is 4 byte. Thus, in a warp it is 4*32=128 byte. //we need to change the struct of haplotype

		float result_block=constant[5];
		char4 read_base_4;
		int i;
		
		char4 read_haplotype_base;
		//if number_address.read_number is even
		for(i=0;i<read_number/2;i++)
		{	
		//got read_base from globle memory  (which is 32*4 (char4) = 128 bytes )
		if(i%2==0)
		read_base_4=read_base_array[i/2*constant_int[2]];
		if(i%2==0)
                {
                read_haplotype_base.x=read_base_4.x;
                read_haplotype_base.z=read_base_4.y;
                }
                if(i%2==1)
                {
                read_haplotype_base.x=read_base_4.z;
                read_haplotype_base.z=read_base_4.w;
                }

		int skip=i*2*constant_int[2];
		parameter1[threadIdx.x]=parameter1_array[skip];
		parameter2[threadIdx.x]=parameter2_array[skip];
		parameter3[threadIdx.x]=parameter3_array[skip];
		parameter4[threadIdx.x]=parameter4_array[skip];
		skip+=constant_int[2];
		parameter1[threadIdx.x+blockDim.x]=parameter1_array[skip];
		parameter2[threadIdx.x+blockDim.x]=parameter2_array[skip];
		parameter3[threadIdx.x+blockDim.x]=parameter3_array[skip];
		parameter4[threadIdx.x+blockDim.x]=parameter4_array[skip];
	
		float Ml=constant[5];// left M;
		float Dl=constant[5];// left D;
		float Il=constant[5];// left I
		float M2=constant[5]; //left M2
		float D2=constant[5]; //left D2

		float MMID=constant[5];
		float MMID2=constant[5];
		float MU=constant[5];// up M;
		float IU=constant[5];// up I;
		float DU=constant[5];// up D;
		
		int hh=(haplotype_number+3)/4;
		for(int j=0;j<hh;j++)
		{
		char4 haplotype_base;
		haplotype_base=haplotype_base_array[j*constant_int[2]]; 
		
		for(int kk=0;kk<4;kk++)
		{
		
			if(j*4+kk==haplotype_number)
				break;
			if(kk==0)
                             read_haplotype_base.y=haplotype_base.x;
                        if(kk==1)
                             read_haplotype_base.y=haplotype_base.y;
                        if(kk==2)
                             read_haplotype_base.y=haplotype_base.z;
                        if(kk==3)
                             read_haplotype_base.y=haplotype_base.w;
				
			int index=(j*4+kk)*blockDim.x*gridDim.x;
			if(i>0)
			{
			//here should not using offset. But using the 
					//get MU,IU,DU from global memory
				MU=MG[index];
				IU=IG[index];
				DU=DG[index];
			}
			else
			{
				DU= constant[0]  /(float)haplotype_number;
				MMID=__fmul_rn(constant[3],DU);
			}
		
			float Qm,Qm_1,alpha,delta,xiksi;
			Qm=parameter1[threadIdx.x];	
			Qm_1=constant[1]-Qm;
			delta=parameter2[threadIdx.x];
			xiksi=parameter3[threadIdx.x];
			alpha=parameter4[threadIdx.x];
			Qm=fdividef(Qm,constant[2]);
		
			float MID=__fadd_rn(IU,DU);
			float DDM=__fmul_rn(Ml,xiksi);
			float IIMI=__fmul_rn(IU,constant[4]);
		
			float aa=(read_haplotype_base.y==read_haplotype_base.x)? Qm_1:Qm;
			float MIIDD=__fmul_rn(constant[3],MID);
			Ml=__fmul_rn(aa,MMID);
			Dl=__fmaf_rn(Dl,constant[4],DDM);
			Il=__fmaf_rn(MU,delta,IIMI);
			MMID=__fmaf_rn(alpha,MU,MIIDD);
	
			skip=threadIdx.x+blockDim.x;	
			Qm=parameter1[skip];	
			Qm_1=constant[1]-Qm;
			delta=parameter2[skip];
			Qm=fdividef(Qm,constant[2]);
			xiksi=parameter3[skip];
			alpha=parameter4[skip];
		
			 MID=__fadd_rn(Il,Dl);
			 DDM=__fmul_rn(M2,xiksi);
			 IIMI=__fmul_rn(Il,constant[4]);
			 aa=(read_haplotype_base.y==read_haplotype_base.z)?Qm_1:Qm;
			 MIIDD=__fmul_rn(constant[3], MID);	

			 M2=__fmul_rn(aa,MMID2);
			 D2=__fmaf_rn(D2,constant[4],DDM);
			 Il=__fmaf_rn(Ml,delta,IIMI);
			 MMID2=__fmaf_rn(alpha,Ml,MIIDD);
				
			if(i==read_number/2-1 && read_number%2==0)
			{
			result_block=__fadd_rn(result_block,__fadd_rn(M2,Il));
			}
			else
			{
				MG[index]=M2;
				IG[index]=Il;
				DG[index]=D2;
			}
		}//4
		}//haplotype
		}//read /2

		//following is only  48 registers
		for(i=i*2;i<read_number;i++)
                {
                
		if(i%4==0)
                {
                        read_base_4=read_base_array[i/4*constant_int[2]];
                }
                
		if(i%4==0) read_haplotype_base.x=read_base_4.x;
                if(i%4==1) read_haplotype_base.x=read_base_4.y;
                if(i%4==2) read_haplotype_base.x=read_base_4.z;
                if(i%4==3) read_haplotype_base.x=read_base_4.w;

                float Qm,Qm_1,alpha,delta,xiksi;
        		
		Qm=parameter1_array[i*constant_int[2]];	
		delta=parameter2_array[i*constant_int[2]];
		Qm_1=constant[1]-Qm;
		xiksi=parameter3_array[i*constant_int[2]];
		alpha=parameter4_array[i*constant_int[2]];
		Qm=fdividef(Qm,constant[2]);

                float Ml=0;// left M;
                float Dl=0;// left D;
                float Il=0;
                float MU=0;// up M;
                float IU=0;// up I;
                float DU=0;// up D;
                float MMID=0;

		 if(i==0)
                {
                DU=constant[0]/(float) haplotype_number;
                MMID=__fmul_rn(constant[3],DU);
                }

                int hh=(haplotype_number+4-1)/4;
                for(int j=0;j<hh;j++)
                {
                char4 haplotype_base;
                haplotype_base=haplotype_base_array[j*constant_int[2]];

                for(int kk=0;kk<4;kk++)
                {
                                if(j*4+kk==haplotype_number)
                                        break;

                                int index=(j*4+kk)*blockDim.x*gridDim.x;
                                if(i>0)
                                {
                                        //here should not using offset. But using the
                                        //get MU,IU,DU from global memory
                                        MU=MG[index];
                                        IU=IG[index];
                                        DU=DG[index];
                                }

                                float MID=__fadd_rn(IU,DU);
                                float DDM=__fmul_rn(Ml,xiksi);
                                float IIMI=__fmul_rn(IU,constant[4]);
                                if(kk==0)
                                       read_haplotype_base.y=haplotype_base.x;
                                if(kk==1)
                                        read_haplotype_base.y=haplotype_base.y;

				 if(kk==2)
                                        read_haplotype_base.y=haplotype_base.z;
                                if(kk==3)
                                        read_haplotype_base.y=haplotype_base.w;

                                float aa=(read_haplotype_base.y==read_haplotype_base.x)? Qm_1:Qm;

                                float MIIDD=__fmul_rn(constant[3],MID);
                                Ml=__fmul_rn(aa,MMID);
                                Il=__fmaf_rn(MU,delta,IIMI);
                                Dl=__fmaf_rn(Dl,constant[4],DDM);

                                MMID=__fmaf_rn(alpha,MU,MIIDD);

                                if(i<read_number-1)
                                {
                                MG[index]=Ml;
                                IG[index]=Il;
                                DG[index]=Dl;
                                }
                                else
                                        result_block=__fadd_rn(result_block,__fadd_rn(Ml,Il));
                        }//4
                } //haplotype

                }//read

		result[offset]=result_block;
		offset+=gridDim.x*blockDim.x ;	
	 }

}


struct InputData
{
int read_size;
char read_base[260];
char base_quals[260];
char ins_quals[260];
char del_quals[260];
char gcp_quals[260];
int haplotype_size;
char haplotype_base[500];
};

bool operator<(const InputData &a, const InputData &b)
{
 //   return x.point_value > y.point_value;
        if(a.read_size<b.read_size) return true;
        if(a.read_size==b.read_size) return a.haplotype_size<b.haplotype_size;
        else
        return false;

}




int main(int argc, char * argv[])
{
		int INI=(log10f((std::numeric_limits<float>::max() / 16)));
		//printf("input value of size_each_for \n");
		//scanf("%d", &size_each_for);
		struct timespec start,finish;
		double  computation_time=0,mem_cpy_time=0,read_time=0, data_prepare=0;
		double total_time=0;
		float * MG;
		float * DG;
		float * IG;
		hipMalloc( (float **)& MG,sizeof(float) *128*105*500*3);
		DG=MG+128*105*500;// ????
		IG=DG+128*105*500;  //?????

		FILE * file;
	//	file=fopen("pairHMM_input_store.txt","r");
	//	file=fopen(argv[1],"r");
		//file=fopen("32_data.txt","r");
	//	file=fopen("less.txt","r");
	file=fopen("/data/04068/sren/dir_chromosome-10/b.txt","r");
	//	 file=fopen("../a.txt","r");
		int size;
		fscanf(file,"%d",&size);

		clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
		float ph2pr_h[128];
		for(int i=0;i<128;i++)
		{
			ph2pr_h[i]=powf(10.f, -((float)i) / 10.f);
		}
		hipError_t err;
		
		int  constants_h_int[10];
		float constants_h[10];
		constants_h[0]=1.329228e+36;
		constants_h[1]=1.0;
		constants_h[2]=3.0;
		constants_h[3]=0.9;
		constants_h[4]=0.1;
		constants_h[5]=0.0;
		constants_h_int[0]=0;
		constants_h_int[1]=128;
		constants_h_int[2]=32;
		constants_h_int[3]=4;
		constants_h_int[4]=3;

		hipMemcpyToSymbol(HIP_SYMBOL(constant),constants_h,sizeof(float)*10 );
		hipMemcpyToSymbol(HIP_SYMBOL(constant_int),constants_h_int,sizeof(int)*10 );
			
	
		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);	
		data_prepare+=diff(start,finish);
		
		int total=0;
		char * result_d_total;
		float read_read, haplotype_haplotype;
		while(!feof(file))
		{
			total+=size;
			char useless;
			useless=fgetc(file);
			
			clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
			
			InputData *inputdata=(InputData* )malloc(size*(sizeof(InputData)));		
			for(int i=0;i<size;i++)
			{
				int read_size;
				fscanf(file,"%d\n",&inputdata[i].read_size);
				fscanf(file,"%s ",inputdata[i].read_base);
				read_size=inputdata[i].read_size;
				read_read=read_size;
			
				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i]. base_quals[j]=(char)aa;
				}

				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i].ins_quals[j]=(char)aa;
				}
				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i].del_quals[j]=(char)aa;
				}

				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				if(j<read_size-1) fscanf(file,"%d ",&aa);
				else  fscanf(file,"%d \n",&aa);
				 inputdata[i].gcp_quals[j]=(char)aa;
				}

				fscanf(file,"%d\n",&inputdata[i].haplotype_size);
				fscanf(file, "%s\n",inputdata[i].haplotype_base);
				haplotype_haplotype=inputdata[i].haplotype_size;
			}
			clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
			read_time+=diff(start,finish);
			
   			float * result_h=(float *) malloc(sizeof(float)*size);
		
			struct timespec start_total,finish_total;
		
			clock_gettime(CLOCK_MONOTONIC_RAW,&start_total); 
			char * data_h_total;

			   std::sort(inputdata, inputdata+size);	
			//32 one chunck.
			int malloc_size_for_each_chunk=(65*4*32+260*4*32*4+125*4*32) ;
			int total_size=(size+31)/32*malloc_size_for_each_chunk+(size*sizeof(NUM_ADD)+127)/128*128;
			data_h_total=(char*)malloc(total_size);
		
			err=hipMalloc( (char **) &result_d_total,total_size+size*sizeof(float));
        		if(err!=hipSuccess)
                        printf("Error %d:%s !\n", err, hipGetErrorString(err));
			char * data_d_total=result_d_total;
	             	float * result_d=(float *)(result_d_total+total_size);//last part is to store the result.     

			char * data_h=data_h_total;
			char * data_h_begin=data_h; 
			NUM_ADD *data_num_add=(NUM_ADD *) (data_h);
			
			data_h=data_h+(size*sizeof(NUM_ADD)+127)/128*128; // it is 64*x .thus we donot need to worry about alignment.
			int data_size=0;
		
			//for each chunk
			int total_in_each=(size+31)/32;
			for(int i=0;i<total_in_each;i++)
			{
			//each is 32 
			//printf("total_in_each %d\n",total_in_each);
			//read_base
			int long_read_size=0;
			//to find the longest read_size
			for(int j=0;j<32;j++)
			{
			if(i*32+j>=size)
				break;
			if(long_read_size<inputdata[i*32+j].read_size)
				long_read_size=inputdata[i*32+j].read_size;
			}

			int change_length=(long_read_size+3)/4;//because tile=4; each time deal with 4 read
			char4 read_base_data[32*65];
			for(int kk=0;kk<change_length;kk++)
			{
				for(int dd=0;dd<32;dd++) //
				{
					if(i*32+dd>=size)
						break;

					if(inputdata[i*32+dd].read_size<=kk*4)
						continue;
					else
					read_base_data[kk*32+dd].x=inputdata[i*32+dd].read_base[kk*4];
				
					if(inputdata[i*32+dd].read_size<=kk*4+1)
						continue;
					else
					read_base_data[kk*32+dd].y=inputdata[i*32+dd].read_base[kk*4+1];
					
					if(inputdata[i*32+dd].read_size<=kk*4+2)
						continue;
					else
					read_base_data[kk*32+dd].z=inputdata[i*32+dd].read_base[kk*4+2];
				
					if(inputdata[i*32+dd].read_size<=kk*4+3)
						continue;
					else
					read_base_data[kk*32+dd].w=inputdata[i*32+dd].read_base[kk*4+3];
				}
			}	
			//finish read_base

			float parameter1[260*32];//Qm//128 do not change to 128
			float parameter2[260*32];//QI//128 do not change to 128
			float parameter3[260*32];//QD/128 do not change to 128
			float parameter4[260*32];//alpha//128 do not change to 128
			for(int kk=0;kk<long_read_size;kk++)
			{
				for(int dd=0;dd<32;dd++)
				{
					if(i*32+dd>=size)
						break;
					
					if(inputdata[i*32+dd].read_size<=kk)
						continue;
					else
					{
					parameter1[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].base_quals[kk]&127];   
					parameter2[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].ins_quals[kk]&127]  ;
					parameter3[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].del_quals[kk]&127] ;
					parameter4[kk*32+dd]= 1.0f-ph2pr_h[((int)(inputdata[i*32+dd].ins_quals[kk]&127)+(int)( inputdata[i*32+dd].del_quals[kk]&127))&127];
		//			printf("kk=%d  x=%d  y=%d z=%d w=%d \n ",kk,parameter1[kk*32+dd],parameter2[kk*32+dd],parameter3[kk*32+dd],parameter4[kk*32+dd] );
					}		
				}
			}
			
			//to haplotype into 32 char4
			int long_haplotype_size=0;
			//to find the longest hapltoype_size
			for(int j=0;j<32;j++)
			{
			if(i*32+j>=size)
				break;
			if(long_haplotype_size<inputdata[i*32+j].haplotype_size)
				long_haplotype_size=inputdata[i*32+j].haplotype_size;
			}

			int haplotype_change_length=(long_haplotype_size+3)/4;
			char4 haplotype_base_data[125*32];
			for(int kk=0;kk<haplotype_change_length;kk++)
			{
				for(int dd=0;dd<32;dd++)
				{
					if(i*32+dd>=size)
						break;
					if(inputdata[i*32+dd].haplotype_size<=kk*4)
						continue;
					else
					haplotype_base_data[kk*32+dd].x=inputdata[i*32+dd].haplotype_base[kk*4];
				
					if(inputdata[i*32+dd].haplotype_size<=kk*4+1)
						continue;
					else
					haplotype_base_data[kk*32+dd].y=inputdata[i*32+dd].haplotype_base[kk*4+1];
					
					if(inputdata[i*32+dd].haplotype_size<=kk*4+2)
						continue;
					else
					haplotype_base_data[kk*32+dd].z=inputdata[i*32+dd].haplotype_base[kk*4+2];
				
					if(inputdata[i*32+dd].haplotype_size<=kk*4+3)
						continue;
					else
					haplotype_base_data[kk*32+dd].w=inputdata[i*32+dd].haplotype_base[kk*4+3];
				}
			}

			//put data address to each pair of read and haplotype.
			// read address
			memcpy(data_h,read_base_data,sizeof(char4)*32*change_length);//128
			for(int kk=0;kk<32;kk++)
			{
				if(i*32+kk>=size) break;
				data_num_add[i*32+kk].read_number=inputdata[i*32+kk].read_size;
				data_num_add[i*32+kk].haplotype_number=inputdata[i*32+kk].haplotype_size;
				data_num_add[i*32+kk].Read_array=data_size+sizeof(char4)*kk;
		//		printf("set read size %d %d \n", data_num_add[i*32+kk].read_number,data_num_add[i*32+kk].haplotype_number);
			}

			data_h+=sizeof(char4)*32*change_length;
			data_size+=sizeof(char4)*32*change_length;
			
			//parameter address
			memcpy(data_h,parameter1,sizeof(float)*32*long_read_size);
			for(int kk=0;kk<32;kk++)
			{
				if(i*32+kk>=size) break;
				data_num_add[i*32+kk].read_large_length=long_read_size;
			}
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
			
			memcpy(data_h,parameter2,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		
			memcpy(data_h,parameter3,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		
			memcpy(data_h,parameter4,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		

			//haplotype address
			memcpy(data_h,haplotype_base_data,sizeof(char4)*32*haplotype_change_length);
			data_h+=sizeof(char4)*32*haplotype_change_length;
			data_size+=sizeof(char4)*32*haplotype_change_length;
			}
				
			int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+127)/128*128;			
			char * data_d;
			NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
			data_d=data_d_total+(sizeof(NUM_ADD)*size+127)/128*128;
			//printf("data_d_total  %p   num_add_d  %p     data_d %p \n",data_d_total,  num_add_d,data_d);		
			int blocksize=128;
			int gridsize=105;
			dim3 block(blocksize);
			dim3 grid(gridsize);
			// global memory to be used by GPU kernels.
	 	
			clock_gettime(CLOCK_MONOTONIC_RAW,&start);
			err=hipMemcpy(data_d_total,data_h_begin,data_size_to_copy,hipMemcpyHostToDevice);
			if(err!=hipSuccess)
			printf("Error %d: %s !\n", err, hipGetErrorString(err));
			pairHMM<<<grid,block>>> (size,data_d,num_add_d, result_d,MG,DG,IG);
                        hipMemcpy(result_h,result_d,size*sizeof(float),hipMemcpyDeviceToHost);
                       	clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
    			computation_time+=diff(start,finish);
		    	for(int i=0;i<size;i++)
float aa=(log10f((double)result_h[i]) - INI);
		//	   	printf("  i=%d  %e\n",i, result_h[i]);
		
			free(result_h);
			free(data_h_total);
         		hipFree(result_d_total);
	//		
			
                       	clock_gettime(CLOCK_MONOTONIC_RAW,&finish_total);
			total_time+=diff(start_total,finish_total);		
			free(inputdata);
			fscanf(file,"%d",&size);
	//	if(total>10000)
	//		break;
		}
		
		clock_gettime(CLOCK_MONOTONIC_RAW,&start);
 		hipFree(MG);   
	 	hipDeviceReset();
		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
		mem_cpy_time+=diff(start,finish);//(finish1.tv_nsec-start1.tv_nsec)/1000000000.0;

		printf("read_time=%e  initial_time=%e  computation_time= %e total_time=%e\n",read_time, data_prepare,computation_time, total_time);
		//printf("GCUPS: %lf \n",  fakesize*read_read*haplotype_haplotype/computation_time/1000000000);
		printf("Total time=%e\n",total_time);
		return 0;
	}


