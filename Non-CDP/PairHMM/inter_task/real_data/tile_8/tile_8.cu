	#include <iostream>
	#include<limits>
	#include <stdlib.h>
	#include <stdio.h>
	#include <string.h>
	#include <time.h>
	#include <hip/hip_runtime.h>
	#include <stdint.h>
	#include <math.h>
	#include <unistd.h>
	#include <omp.h>	
	#include <algorithm>
	using namespace std;

	// 8 byte.   how to be 128byte?
	// Parameter need to restruct.
	//2 bytes, 2 bytes, 4 bytes, 4 bytes, 4 bytes.
	struct NUM_ADD
	{
		short2 read_haplotype;
		int  Read_array;
		int read_large_length;
	};

	double diff(timespec start, timespec end)
	{
	  double a=0;
	 if((end.tv_nsec-start.tv_nsec)<0)
	{
	a=end.tv_sec-start.tv_sec-1;
	a+=(1000000000+end.tv_nsec-start.tv_nsec)/1000000000.0;
	}
	else
	{
	a=end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

	}
	return a;
	}

	__constant__ float  constant[10];
	__constant__ int  constant_int[10];
	

	__global__ void  pairHMM( int size, char * data,  NUM_ADD * num_add, float * result,float * MG,float * DG, float * IG ) // what is the maximum number of parameters?
	{
	//MG, DG and IG are global memory to store indermediate result?
	//each thread finish one computation		
	int offset=blockIdx.x*blockDim.x+threadIdx.x;
	MG=MG+offset;
	IG=IG+offset;
	DG=DG+offset;
	//printf("%d %d %d %d %d\n", constant_int[0],constant_int[1], constant_int[2],constant_int[3], constant_int[4]);
	while(offset<size)
	 {	
		__shared__ float parameter1[1024];
		__shared__ float parameter2[1024];
		__shared__ float parameter3[1024];
		__shared__ float parameter4[1024];
	
		//NUM_ADD number_address;
		//number_address=num_add[offset];//get from global memory
		
		short2 read_haplotype_number=num_add[offset].read_haplotype;
		int read_large_length=num_add[offset].read_large_length;
		//read_haplotype_number.x=number_address.read_number;	
		char4 * read_base_array=(char4 *)(data+num_add[offset].Read_array); // to caculate the address of read_base_array. 
		float  *parameter1_array=(float *) (read_base_array+(read_large_length+3)/4*32);
		read_large_length=read_large_length*32;
		float  *parameter2_array=(float *) (parameter1_array+read_large_length);
		float  *parameter3_array=(float *) (parameter1_array+read_large_length*2);
		float  *parameter4_array=(float *) (parameter1_array+read_large_length*3);
		//read_haplotype_number.y=number_address.haplotype_number;
		char4 * haplotype_base_array=(char4 * )(parameter1_array+read_large_length*4);    	
		//haplotype is 4 byte. Thus, in a warp it is 4*32=128 byte. //we need to change the struct of haplotype

		float result_block=constant[5];
		char4 read_base_4_1;
		char4 read_base_4_2;
		int i;
		
		//if number_address.read_number is even
		
		//int time=0;
	//if(threadIdx.x==0)
		for(i=0;i<read_haplotype_number.x/8;i++)
		{	
		//got read_base from globle memory  (which is 32*4 (char4) = 128 bytes )
		//read_base_4=read_base_array[i*constant_int[2]];

		char4 read_base_temp;
		int cc=i*2*32;
		read_base_temp=read_base_array[cc];
		read_base_4_1.x=read_base_temp.x;
		read_base_4_1.y=read_base_temp.y;
		read_base_4_1.z=read_base_temp.z;
		read_base_4_1.w=read_base_temp.w;

		read_base_temp=read_base_array[cc+32];
		read_base_4_2.x=read_base_temp.x;
		read_base_4_2.y=read_base_temp.y;
		read_base_4_2.z=read_base_temp.z;
		read_base_4_2.w=read_base_temp.w;



		int skip=i*constant_int[1];
		parameter1[threadIdx.x]=parameter1_array[skip];
		parameter2[threadIdx.x]=parameter2_array[skip];
		parameter3[threadIdx.x]=parameter3_array[skip];
		parameter4[threadIdx.x]=parameter4_array[skip];
		skip+=constant_int[2];
		parameter1[threadIdx.x+blockDim.x]=parameter1_array[skip];
		parameter2[threadIdx.x+blockDim.x]=parameter2_array[skip];
		parameter3[threadIdx.x+blockDim.x]=parameter3_array[skip];
		parameter4[threadIdx.x+blockDim.x]=parameter4_array[skip];
		skip+=constant_int[2];
		parameter1[threadIdx.x+blockDim.x*2]=parameter1_array[skip];
		parameter2[threadIdx.x+blockDim.x*2]=parameter2_array[skip];
		parameter3[threadIdx.x+blockDim.x*2]=parameter3_array[skip];
		parameter4[threadIdx.x+blockDim.x*2]=parameter4_array[skip];
		skip+=constant_int[2];
		parameter1[threadIdx.x+blockDim.x*3]=parameter1_array[skip];
		parameter2[threadIdx.x+blockDim.x*3]=parameter2_array[skip];
		parameter3[threadIdx.x+blockDim.x*3]=parameter3_array[skip];
		parameter4[threadIdx.x+blockDim.x*3]=parameter4_array[skip];
		skip+=constant_int[2];
		parameter1[threadIdx.x+blockDim.x*4]=parameter1_array[skip];
		parameter2[threadIdx.x+blockDim.x*4]=parameter2_array[skip];
		parameter3[threadIdx.x+blockDim.x*4]=parameter3_array[skip];
		parameter4[threadIdx.x+blockDim.x*4]=parameter4_array[skip];
		skip+=constant_int[2];
		parameter1[threadIdx.x+blockDim.x*5]=parameter1_array[skip];
		parameter2[threadIdx.x+blockDim.x*5]=parameter2_array[skip];
		parameter3[threadIdx.x+blockDim.x*5]=parameter3_array[skip];
		parameter4[threadIdx.x+blockDim.x*5]=parameter4_array[skip];

	    	skip+=constant_int[2];
		parameter1[threadIdx.x+blockDim.x*6]=parameter1_array[skip];
		parameter2[threadIdx.x+blockDim.x*6]=parameter2_array[skip];
		parameter3[threadIdx.x+blockDim.x*6]=parameter3_array[skip];
		parameter4[threadIdx.x+blockDim.x*6]=parameter4_array[skip];
		
		skip+=constant_int[2];
		parameter1[threadIdx.x+blockDim.x*7]=parameter1_array[skip];
		parameter2[threadIdx.x+blockDim.x*7]=parameter2_array[skip];
		parameter3[threadIdx.x+blockDim.x*7]=parameter3_array[skip];
		parameter4[threadIdx.x+blockDim.x*7]=parameter4_array[skip];


		float Ml=constant[5];// left M;
		float Dl=constant[5];// left D;
		float Il=constant[5];// left I
		float M2=constant[5]; //left M2
		float D2=constant[5]; //left D2
		float M3=constant[5];
		float D3=constant[5];
		float M4=constant[5];
		float D4=constant[5];

		float M5=0;
		float D5=0;
		float M6=0;
		float D6=0;
		float M7=0;
		float D7=0;
		float M8=0;
		float D8=0;

		float MU=constant[5];// up M;
		float IU=constant[5];// up I;
		float DU=constant[5];// up D;
		float MMID=constant[5];
		float MMID2=constant[5];
		float MMID3=constant[5];
		float MMID4=constant[5];
		float MMID5=constant[5];
		float MMID6=constant[5];
		float MMID7=constant[5];
		float MMID8=constant[5];
		//epsion=constant[4];			
	//	beta=constant[3];
		
		int hh=(read_haplotype_number.y+3)/4;
		
		for(int j=0;j<hh;j++)
		{
		char4 haplotype_base;
		haplotype_base=haplotype_base_array[j*constant_int[2]]; 
		
		for(int kk=0;kk<4;kk++)
		{
			//time++;
			float Qm,Qm_1,alpha,delta,xiksi;
			if(j*4+kk==read_haplotype_number.y)
				break;
				
			int index=(j*4+kk)*blockDim.x*gridDim.x;
			if(i>0)
			{
			//here should not using offset. But using the 
					//get MU,IU,DU from global memory
				MU=MG[index];
				IU=IG[index];
				DU=DG[index];
			}

			else
			{
				DU= constant[0]  /(float) read_haplotype_number.y;
				MMID=__fmul_rn(constant[3],DU);
			}
		
			Qm=parameter1[threadIdx.x];	
			delta=parameter2[threadIdx.x];
			xiksi=parameter3[threadIdx.x];
			alpha=parameter4[threadIdx.x];
			Qm_1=constant[1]-Qm;
			Qm=fdividef(Qm,constant[2]);
		

			float MID=__fadd_rn(IU,DU);
			float DDM=__fmul_rn(Ml,xiksi);
			float IIMI=__fmul_rn(IU,constant[4]);
		//	if(i==1)printf("%e %e", IIMI, MU);
                        char4 read_haplotype_base;
			if(kk==0)
                             read_haplotype_base.y=haplotype_base.x;
                        if(kk==1)
                             read_haplotype_base.y=haplotype_base.y;
                        if(kk==2)
                             read_haplotype_base.y=haplotype_base.z;
                        if(kk==3)
                             read_haplotype_base.y=haplotype_base.w;
			
			float aa=(read_haplotype_base.y==read_base_4_1.x)? Qm_1:Qm;
			float MIIDD=__fmul_rn(constant[3],MID);
			Ml=__fmul_rn(aa,MMID);
			Dl=__fmaf_rn(Dl,constant[4],DDM);
			Il=__fmaf_rn(MU,delta,IIMI);
			MMID=__fmaf_rn(alpha,MU,MIIDD);
	//2
//	if(i==1)             printf("R=%c H=%c  M1=%e I1=%e  D1=%e\n", read_base_4_1.x,read_haplotype_base.y,Ml,Il,Dl);
			skip=threadIdx.x+blockDim.x;	
			Qm=parameter1[skip];	
			delta=parameter2[skip];
			xiksi=parameter3[skip];
			alpha=parameter4[skip];
			Qm_1=constant[1]-Qm;
			Qm=fdividef(Qm,constant[2]);
//	if(i==1) printf("%e %e %e %e %e", Qm,delta,xiksi,alpha,Qm_1);	
			 MID=__fadd_rn(Il,Dl);
			 DDM=__fmul_rn(M2,xiksi);
			 IIMI=__fmul_rn(Il,constant[4]);
			 aa=(read_haplotype_base.y==read_base_4_1.y)?Qm_1:Qm;
			 MIIDD=__fmul_rn(constant[3], MID);	

			 M2=__fmul_rn(aa,MMID2);
			 D2=__fmaf_rn(D2,constant[4],DDM);
			 Il=__fmaf_rn(Ml,delta,IIMI);
			 MMID2=__fmaf_rn(alpha,Ml,MIIDD);
	
//	if(i==1)             printf("R=%c H=%c  M1=%e I1=%e  D1=%e\n", read_base_4_1.y,read_haplotype_base.y,M2,Il,D2);
		//3
			skip+=blockDim.x;
			Qm=parameter1[skip];	
			delta=parameter2[skip];
			xiksi=parameter3[skip];
			alpha=parameter4[skip];
			//epsion=0.1;			
			//beta=1.0-epsion;
			Qm_1=constant[1]-Qm;
			Qm=fdividef(Qm,constant[2]);

			 MID=__fadd_rn(Il,D2);
			 DDM=__fmul_rn(M3,xiksi);
			 IIMI=__fmul_rn(Il,constant[4]);
			 aa=(read_haplotype_base.y==read_base_4_1.z)?Qm_1:Qm;
			 MIIDD=__fmul_rn(constant[3], MID);	

			 M3=__fmul_rn(aa,MMID3);
			 D3=__fmaf_rn(D3,constant[4],DDM);
			 Il=__fmaf_rn(M2,delta,IIMI);
			 MMID3=__fmaf_rn(alpha,M2,MIIDD);
	
//	if(i==1)             printf("R=%c H=%c  M3=%e I3=%e  D3=%e\n", read_base_4_1.z,read_haplotype_base.y,M3,Il,D3);
		//4
			skip+=blockDim.x;
			Qm=parameter1[skip];	
			delta=parameter2[skip];
			xiksi=parameter3[skip];
			alpha=parameter4[skip];
			Qm_1=constant[1]-Qm;
			Qm=fdividef(Qm,constant[2]);

			 MID=__fadd_rn(Il,D3);
			 DDM=__fmul_rn(M4,xiksi);
			 IIMI=__fmul_rn(Il,constant[4]);
			 aa=(read_haplotype_base.y==read_base_4_1.w)?Qm_1:Qm;
			 MIIDD=__fmul_rn(constant[3], MID);	

			 M4=__fmul_rn(aa,MMID4);
			 D4=__fmaf_rn(D4,constant[4],DDM);
			 Il=__fmaf_rn(M3,delta,IIMI);
			 MMID4=__fmaf_rn(alpha,M3,MIIDD);
	
//	if(i==1)             printf("R=%c H=%c  M4=%e I4=%e  D4=%e\n", read_base_4_2.y,read_haplotype_base.y,M4,Il,D4);
	//5
			skip+=blockDim.x;
			Qm=parameter1[skip];	
			delta=parameter2[skip];
			xiksi=parameter3[skip];
			alpha=parameter4[skip];
			Qm_1=constant[1]-Qm;
			Qm=fdividef(Qm,constant[2]);

			 MID=__fadd_rn(Il,D4);
			 DDM=__fmul_rn(M5,xiksi);
			 IIMI=__fmul_rn(Il,constant[4]);
			 aa=(read_haplotype_base.y==read_base_4_2.x)?Qm_1:Qm;
			 MIIDD=__fmul_rn(constant[3], MID);	

			 M5=__fmul_rn(aa,MMID5);
			 D5=__fmaf_rn(D5,constant[4],DDM);
			 Il=__fmaf_rn(M4,delta,IIMI);
			 MMID5=__fmaf_rn(alpha,M4,MIIDD);

// if(i==1)
  //                      printf("R=%c H=%c  M5=%e I5=%e  D5=%e\n", read_base_4_2.y,read_haplotype_base.y,M5,Il,D5);
			//6
			skip+=blockDim.x;
			Qm=parameter1[skip];	
			delta=parameter2[skip];
			xiksi=parameter3[skip];
			alpha=parameter4[skip];
			Qm_1=constant[1]-Qm;
			Qm=fdividef(Qm,constant[2]);

			 MID=__fadd_rn(Il,D5);
			 DDM=__fmul_rn(M6,xiksi);
			 IIMI=__fmul_rn(Il,constant[4]);
			 aa=(read_haplotype_base.y==read_base_4_2.y)?Qm_1:Qm;
			 MIIDD=__fmul_rn(constant[3], MID);	

			 M6=__fmul_rn(aa,MMID6);
			 D6=__fmaf_rn(D6,constant[4],DDM);
			 Il=__fmaf_rn(M5,delta,IIMI);
			 MMID6=__fmaf_rn(alpha,M5,MIIDD);

	// if(i==1) printf("R=%c H=%c  M6=%e I6=%e  D6=%e\n", read_base_4_2.y,read_haplotype_base.y,M6,Il,D6);
	//7
	
		        skip+=blockDim.x;
			Qm=parameter1[skip];	
			delta=parameter2[skip];
			xiksi=parameter3[skip];
			alpha=parameter4[skip];
			Qm_1=constant[1]-Qm;
			Qm=fdividef(Qm,constant[2]);

			 MID=__fadd_rn(Il,D6);
			 DDM=__fmul_rn(M7,xiksi);
			 IIMI=__fmul_rn(Il,constant[4]);
			 aa=(read_haplotype_base.y==read_base_4_2.z)?Qm_1:Qm;
			 MIIDD=__fmul_rn(constant[3], MID);	

			 M7=__fmul_rn(aa,MMID7);
			 D7=__fmaf_rn(D7,constant[4],DDM);
			 Il=__fmaf_rn(M6,delta,IIMI);
			 MMID7=__fmaf_rn(alpha,M6,MIIDD);
			 //8
			skip+=blockDim.x;
			Qm=parameter1[skip];	
			delta=parameter2[skip];
			xiksi=parameter3[skip];
			alpha=parameter4[skip];
			Qm_1=constant[1]-Qm;
			Qm=fdividef(Qm,constant[2]);

			 MID=__fadd_rn(Il,D7);
			 DDM=__fmul_rn(M8,xiksi);
			 IIMI=__fmul_rn(Il,constant[4]);
			 aa=(read_haplotype_base.y==read_base_4_2.w)?Qm_1:Qm;
			 MIIDD=__fmul_rn(constant[3], MID);	

			 M8=__fmul_rn(aa,MMID8);
			 D8=__fmaf_rn(D8,constant[4],DDM);
			 Il=__fmaf_rn(M7,delta,IIMI);
			 MMID8=__fmaf_rn(alpha,M7,MIIDD);

			if(i==read_haplotype_number.x/8-1 && read_haplotype_number.x%8==0)
			{
			result_block=__fadd_rn(result_block,__fadd_rn(M8,Il));
			}
			else
			{
				MG[index]=M8;
				IG[index]=Il;
				DG[index]=D8;
			}
		}//8

		}//haplotype
		}
	//	if(threadIdx.x==0)
	//	printf("time=%d\n",time);
		//following is only  56 registers
		
		for(i=i*8;i<read_haplotype_number.x;i++)
                {
                //char4 read_base_4;
                if(i%4==0)
                {
                        read_base_4_1=read_base_array[i/4*constant_int[2]];
                }
                char4 read_haplotype_base;
                if(i%4==0) read_haplotype_base.x=read_base_4_1.x;
                if(i%4==1) read_haplotype_base.x=read_base_4_1.y;
                if(i%4==2) read_haplotype_base.x=read_base_4_1.z;
                if(i%4==3) read_haplotype_base.x=read_base_4_1.w;

                float Qm,Qm_1,alpha,delta,xiksi;
        		
		delta=parameter2_array[i*constant_int[2]];
		xiksi=parameter3_array[i*constant_int[2]];
		alpha=parameter4_array[i*constant_int[2]];
		Qm=parameter1_array[i*constant_int[2]];	
		Qm_1=constant[1]-Qm;
		Qm=fdividef(Qm,constant[2]);

                float Ml=0;// left M;
                float Dl=0;// left D;
                float Il=0;
                float MU=0;// up M;
                float IU=0;// up I;
                float DU=0;// up D;
                float MMID=0;

		 if(i==0)
                {
                DU=constant[0]/(float) read_haplotype_number.y;
                MMID=__fmul_rn(constant[3],DU);
                }

                int hh=(read_haplotype_number.y+4-1)/4;
                for(int j=0;j<hh;j++)
                {
                char4 haplotype_base;
                haplotype_base=haplotype_base_array[j*constant_int[2]];

                for(int kk=0;kk<4;kk++)
                {
                                if(j*4+kk==read_haplotype_number.y)
                                        break;
 				 if(kk==0)
                                       read_haplotype_base.y=haplotype_base.x;
                                if(kk==1)
                                        read_haplotype_base.y=haplotype_base.y;

				 if(kk==2)
                                        read_haplotype_base.y=haplotype_base.z;
                                if(kk==3)
                                        read_haplotype_base.y=haplotype_base.w;


                                int index=(j*4+kk)*blockDim.x*gridDim.x;
                                if(i>0)
                                {
                                        //here should not using offset. But using the
                                        //get MU,IU,DU from global memory
                                        MU=MG[index];
                                        IU=IG[index];
                                        DU=DG[index];
                                }

                                float MID=__fadd_rn(IU,DU);
                                float DDM=__fmul_rn(Ml,xiksi);
                                float IIMI=__fmul_rn(IU,constant[4]);
                                float aa=(read_haplotype_base.y==read_haplotype_base.x)? Qm_1:Qm;

                                float MIIDD=__fmul_rn(constant[3],MID);
                                Ml=__fmul_rn(aa,MMID);
                                Il=__fmaf_rn(MU,delta,IIMI);
                                Dl=__fmaf_rn(Dl,constant[4],DDM);

                                MMID=__fmaf_rn(alpha,MU,MIIDD);

                                if(i<read_haplotype_number.x-1)
                                {
                                MG[index]=Ml;
                                IG[index]=Il;
                                DG[index]=Dl;
                                }
                                else
                                        result_block=__fadd_rn(result_block,__fadd_rn(Ml,Il));
                        }//4
                } //haplotype

                }//read

		result[offset]=result_block;
		offset+=gridDim.x*blockDim.x ;	
	 }

}


struct InputData
{
int read_size;
char read_base[260];
char base_quals[260];
char ins_quals[260];
char del_quals[260];
char gcp_quals[260];
int haplotype_size;
char haplotype_base[500];
};

  bool operator<(const InputData &a, const InputData &b)
{
 //   return x.point_value > y.point_value;
        if(a.read_size<b.read_size) return true;
        if(a.read_size==b.read_size) return a.haplotype_size<b.haplotype_size;
        else
        return false;

}



int main(int argc, char * argv[])
{
		int INI=(log10f((std::numeric_limits<float>::max() / 16)));
		//printf("input value of size_each_for \n");
		//scanf("%d", &size_each_for);
		struct timespec start,finish;
		double  computation_time=0,mem_cpy_time=0,read_time=0, data_prepare=0;
		double total_time=0;
		FILE * file;
		file=fopen("/data/04068/sren/dir_chromosome-10/b.txt","r");
	//file=fopen("../a.txt","r");
		//	file=fopen(argv[1],"r");
		//file=fopen("32_data.txt","r");
	//	file=fopen("less.txt","r");
		int size;
		fscanf(file,"%d",&size);
		  float * MG;
                float * DG;
                float * IG;

                hipMalloc( (float **)& MG,sizeof(float) *128*45*500*3);
                DG=MG+45*128*500;// ????
                IG=DG+45*128*500;  //?????

		clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
		float ph2pr_h[128];
		for(int i=0;i<128;i++)
		{
			ph2pr_h[i]=powf(10.f, -((float)i) / 10.f);
		}
		hipError_t err;
		
		int  constants_h_int[10];
		float constants_h[10];
		constants_h[0]=1.329228e+36;
		constants_h[1]=1.0;
		constants_h[2]=3.0;
		constants_h[3]=0.9;
		constants_h[4]=0.1;
		constants_h[5]=0.0;
		constants_h_int[0]=0;
		constants_h_int[1]=32*8;
		constants_h_int[2]=32;
		constants_h_int[3]=4;
		constants_h_int[4]=3;

		hipMemcpyToSymbol(HIP_SYMBOL(constant),constants_h,sizeof(float)*10 );
		hipMemcpyToSymbol(HIP_SYMBOL(constant_int),constants_h_int,sizeof(int)*10 );
			
	
		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);	
	//	data_prepare+=diff(start,finish);
		
		int total=0;
		char * result_d_total;
		float read_read, haplotype_haplotype;
		while(!feof(file))
		{
			total+=size;
			char useless;
			useless=fgetc(file);
			
			clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
			
			InputData *inputdata=(InputData* )malloc(size*(sizeof(InputData)));		
			for(int i=0;i<size;i++)
			{
				int read_size;
				fscanf(file,"%d\n",&inputdata[i].read_size);
				fscanf(file,"%s ",inputdata[i].read_base);
				read_size=inputdata[i].read_size;
				read_read=read_size;
			
				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i]. base_quals[j]=(char)aa;
				}

				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i].ins_quals[j]=(char)aa;
				}
				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i].del_quals[j]=(char)aa;
				}

				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				if(j<read_size-1) fscanf(file,"%d ",&aa);
				else  fscanf(file,"%d \n",&aa);
				 inputdata[i].gcp_quals[j]=(char)aa;
				}

				fscanf(file,"%d\n",&inputdata[i].haplotype_size);
				fscanf(file, "%s\n",inputdata[i].haplotype_base);
				haplotype_haplotype=inputdata[i].haplotype_size;
			}
			clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
			read_time+=diff(start,finish);
			
   			float * result_h=(float *) malloc(sizeof(float)*size);
			
			struct timespec start_total,finish_total;
			clock_gettime(CLOCK_MONOTONIC_RAW,&start_total); 
			char * data_h_total;
					
			

			clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
			std::sort(inputdata, inputdata+size);
				//32 one chunck.
			int malloc_size_for_each_chunk=(65*4*32+260*4*32*4+125*4*32) ;
			int total_size=(size+31)/32*malloc_size_for_each_chunk+(size*sizeof(NUM_ADD)+127)/128*128;
			data_h_total=(char*)malloc(total_size);
		
			err=hipMalloc( (char **) &result_d_total,total_size+size*sizeof(float));
        		if(err!=hipSuccess)
                        printf("Error %d:%s !\n", err, hipGetErrorString(err));
			char * data_d_total=result_d_total;
	             	float * result_d=(float *)(result_d_total+total_size);//last part is to store the result.     

			char * data_h=data_h_total;
			char * data_h_begin=data_h; 
			NUM_ADD *data_num_add=(NUM_ADD *) (data_h);
			
			data_h=data_h+(size*sizeof(NUM_ADD)+127)/128*128; // it is 64*x .thus we donot need to worry about alignment.
			int data_size=0;
		
			//for each chunk
			int total_in_each=(size+31)/32;
			for(int i=0;i<total_in_each;i++)
			{
			//each is 32 
			//printf("total_in_each %d\n",total_in_each);
			//read_base
			int long_read_size=0;
			//to find the longest read_size
			for(int j=0;j<32;j++)
			{
			if(i*32+j>=size)
				break;
			if(long_read_size<inputdata[i*32+j].read_size)
				long_read_size=inputdata[i*32+j].read_size;
			}

			int change_length=(long_read_size+3)/4;//because tile=4; each time deal with 4 read
			char4 read_base_data[32*65];
			for(int kk=0;kk<change_length;kk++)
			{
				for(int dd=0;dd<32;dd++) //
				{
					if(i*32+dd>=size)
						break;

					if(inputdata[i*32+dd].read_size<=kk*4)
						continue;
					else
					read_base_data[kk*32+dd].x=inputdata[i*32+dd].read_base[kk*4];
				
					if(inputdata[i*32+dd].read_size<=kk*4+1)
						continue;
					else
					read_base_data[kk*32+dd].y=inputdata[i*32+dd].read_base[kk*4+1];
					
					if(inputdata[i*32+dd].read_size<=kk*4+2)
						continue;
					else
					read_base_data[kk*32+dd].z=inputdata[i*32+dd].read_base[kk*4+2];
				
					if(inputdata[i*32+dd].read_size<=kk*4+3)
						continue;
					else
					read_base_data[kk*32+dd].w=inputdata[i*32+dd].read_base[kk*4+3];
				}
			}	
			//finish read_base

			float parameter1[260*32];//Qm//128 do not change to 128
			float parameter2[260*32];//QI//128 do not change to 128
			float parameter3[260*32];//QD/128 do not change to 128
			float parameter4[260*32];//alpha//128 do not change to 128
			for(int kk=0;kk<long_read_size;kk++)
			{
				for(int dd=0;dd<32;dd++)
				{
					if(i*32+dd>=size)
						break;
					
					if(inputdata[i*32+dd].read_size<=kk)
						continue;
					else
					{
					parameter1[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].base_quals[kk]&127];   
					parameter2[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].ins_quals[kk]&127]  ;
					parameter3[kk*32+dd]= ph2pr_h[inputdata[i*32+dd].del_quals[kk]&127] ;
					parameter4[kk*32+dd]= 1.0f-ph2pr_h[((int)(inputdata[i*32+dd].ins_quals[kk]&127)+(int)( inputdata[i*32+dd].del_quals[kk]&127))&127];
		//			printf("kk=%d  x=%d  y=%d z=%d w=%d \n ",kk,parameter1[kk*32+dd],parameter2[kk*32+dd],parameter3[kk*32+dd],parameter4[kk*32+dd] );
					}		
				}
			}
			
			//to haplotype into 32 char4
			int long_haplotype_size=0;
			//to find the longest hapltoype_size
			for(int j=0;j<32;j++)
			{
			if(i*32+j>=size)
				break;
			if(long_haplotype_size<inputdata[i*32+j].haplotype_size)
				long_haplotype_size=inputdata[i*32+j].haplotype_size;
			}

			int haplotype_change_length=(long_haplotype_size+3)/4;
			char4 haplotype_base_data[32*125];
			for(int kk=0;kk<haplotype_change_length;kk++)
			{
				for(int dd=0;dd<32;dd++)
				{
					if(i*32+dd>=size)
						break;
					if(inputdata[i*32+dd].haplotype_size<=kk*4)
						continue;
					else
					haplotype_base_data[kk*32+dd].x=inputdata[i*32+dd].haplotype_base[kk*4];
				
					if(inputdata[i*32+dd].haplotype_size<=kk*4+1)
						continue;
					else
					haplotype_base_data[kk*32+dd].y=inputdata[i*32+dd].haplotype_base[kk*4+1];
					
					if(inputdata[i*32+dd].haplotype_size<=kk*4+2)
						continue;
					else
					haplotype_base_data[kk*32+dd].z=inputdata[i*32+dd].haplotype_base[kk*4+2];
				
					if(inputdata[i*32+dd].haplotype_size<=kk*4+3)
						continue;
					else
					haplotype_base_data[kk*32+dd].w=inputdata[i*32+dd].haplotype_base[kk*4+3];
				}
			}

			//put data address to each pair of read and haplotype.
			// read address
			memcpy(data_h,read_base_data,sizeof(char4)*32*change_length);//128
			for(int kk=0;kk<32;kk++)
			{
				if(i*32+kk>=size) break;
				data_num_add[i*32+kk].read_haplotype.x=inputdata[i*32+kk].read_size;
				data_num_add[i*32+kk].read_haplotype.y=inputdata[i*32+kk].haplotype_size;
				data_num_add[i*32+kk].Read_array=data_size+sizeof(char4)*kk;
		//		printf("set read size %d %d \n", data_num_add[i*32+kk].read_number,data_num_add[i*32+kk].haplotype_number);
			}

			data_h+=sizeof(char4)*32*change_length;
			data_size+=sizeof(char4)*32*change_length;
			
			//parameter address
			memcpy(data_h,parameter1,sizeof(float)*32*long_read_size);
			for(int kk=0;kk<32;kk++)
			{
				if(i*32+kk>=size) break;
				data_num_add[i*32+kk].read_large_length=long_read_size;
			}
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
			
			memcpy(data_h,parameter2,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		
			memcpy(data_h,parameter3,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		
			memcpy(data_h,parameter4,sizeof(float)*32*long_read_size);
			data_h+=sizeof(float)*32*long_read_size;
			data_size+=sizeof(float)*32*long_read_size;
		

			//haplotype address
			memcpy(data_h,haplotype_base_data,sizeof(char4)*32*haplotype_change_length);
			data_h+=sizeof(char4)*32*haplotype_change_length;
			data_size+=sizeof(char4)*32*haplotype_change_length;
			}
				
			int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+127)/128*128;			
			char * data_d;
			NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
			data_d=data_d_total+(sizeof(NUM_ADD)*size+127)/128*128;
			//printf("data_d_total  %p   num_add_d  %p     data_d %p \n",data_d_total,  num_add_d,data_d);		
			int blocksize=128;
			int gridsize=45;//90
			dim3 block(blocksize);
			dim3 grid(gridsize);
			// global memory to be used by GPU kernels.
			//float * MG;
			//float * DG;
			//float * IG;

			clock_gettime(CLOCK_MONOTONIC_RAW,&start);
			err=hipMemcpy(data_d_total,data_h_begin,data_size_to_copy,hipMemcpyHostToDevice);
		  	if(err!=hipSuccess)
			printf("Error %d: %s !\n", err, hipGetErrorString(err));
			//cudaMalloc( (float **)& MG,sizeof(float) *blocksize*gridsize*500*3);
			//DG=MG+blocksize*gridsize*500;// ????
			//IG=DG+blocksize*gridsize*500;  //?????
		 	
			
                       	//clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
			//data_prepare+=diff(start,finish);
			pairHMM<<<grid,block>>> (size,data_d,num_add_d, result_d,MG,DG,IG);
		  	hipMemcpy(result_h,result_d,size*sizeof(float),hipMemcpyDeviceToHost);
                       	clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
    			computation_time+=diff(start,finish);
			
		    	for(int i=0;i<size;i++)
		float aa=(log10f((double)result_h[i]) - INI);
	//	   	printf("  i=%d  %e\n",i, result_h[i]);
		
			free(data_h_total);
         		hipFree(result_d_total);
	//		
			
                       	clock_gettime(CLOCK_MONOTONIC_RAW,&finish_total);
			total_time+=diff(start_total,finish_total);		
			

			free(result_h);
			free(inputdata);
			fscanf(file,"%d",&size);
	//	if(total>10000)
	//		break;
	//		printf("%d\n",size);
			}
		
    
			hipFree(MG);	
	 	hipDeviceReset();
		printf("read_time=%e  initial_time=%e  computation_time= %e total_time=%e\n",read_time, data_prepare,computation_time, computation_time+mem_cpy_time);
		printf("total_time=%e\n",total_time);	
	//printf("GCUPS: %lf \n",  fakesize*read_read*haplotype_haplotype/computation_time/1000000000);
		return 0;
	}

