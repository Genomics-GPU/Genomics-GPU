
	#include <iostream>
	#include <stdlib.h>
	#include<limits>
	#include <stdio.h>
	#include <string.h>
	#include <time.h>
	#include <hip/hip_runtime.h>
	#include <stdint.h>
	#include <math.h>
	#include <unistd.h>
	#include <omp.h>	
	using namespace std;

	struct NUM_ADD
	{
		short2 read_haplotype_number;
		int address_array;
	};

	double diff(timespec start, timespec end)
	{
	  double a=0;
	 if((end.tv_nsec-start.tv_nsec)<0)
	{
	a=end.tv_sec-start.tv_sec-1;
	a+=(1000000000+end.tv_nsec-start.tv_nsec)/1000000000.0;
	}
	else
	{
	a=end.tv_sec-start.tv_sec+(end.tv_nsec-start.tv_nsec)/1000000000.0;

	}
	return a;

	}

	__global__ void  pairHMM( int size, char * data,  NUM_ADD * num_add, float * result,float * MG,float * DG, float * IG) // what is the maximum number of parameters?
	{
   	 int warp_index=(blockDim.x*blockIdx.x+threadIdx.x)/32;
    	 int warp_index_in_block=threadIdx.x/32;
    	 int offset=warp_index;
    //printf("threadIdx.x=%d \n", threadIdx.x);
         MG=MG+warp_index*500;
         IG=IG+warp_index*500;
         DG=DG+warp_index*500;

	 while(offset<size)
	 {	
		short2 read_haplotype_number;
		char * read_base_array;
		float * parameter_array;
		char4 * haplotype_base_array; 

		float result_block=0;
		int round;
	        int skip;
		//as each time it will deal with 2 read&haplotype pairs
		// each block deal with one pairs of haplotype & read
		read_haplotype_number=num_add[offset].read_haplotype_number;
		skip=(sizeof(float)*read_haplotype_number.x+127)/128*32;
		read_base_array=(char *) (data+num_add[offset].address_array);
		parameter_array=(float *) (read_base_array+(read_haplotype_number.x+127)/128*128);
		haplotype_base_array=(char4 *) (parameter_array+skip*4);
		int read_number=read_haplotype_number.x;
		int haplotype_number=read_haplotype_number.y;
		
		__shared__ char haplotype_base_in_char[500*128/32];
		
		int hh=(haplotype_number+4-1)/4;
		int thread_in_warp=threadIdx.x-warp_index_in_block*32;
		int tt=(hh+32-1)/32;
		for(int ii=0;ii<tt;ii++)
		{	
			int aa=thread_in_warp+ii*32;
			if(aa< hh)
			{
			char4 haplotype_base_in_thread;
			haplotype_base_in_thread=haplotype_base_array[aa]; //Is it right to get data from global memory
			haplotype_base_in_char[warp_index_in_block*500+aa*4]=haplotype_base_in_thread.x;
			haplotype_base_in_char[warp_index_in_block*500+aa*4+1]=haplotype_base_in_thread.y;
			haplotype_base_in_char[warp_index_in_block*500+aa*4+2]=haplotype_base_in_thread.z;
			haplotype_base_in_char[warp_index_in_block*500+aa*4+3]=haplotype_base_in_thread.w;
			//printf("%c %c %c %c\n", haplotype_base_in_thread.x,haplotype_base_in_thread.y,haplotype_base_in_thread.z, haplotype_base_in_thread.w);
			}
		}
	      //__syncthreads();
		
		float D_0=1.329228e+36/(float)haplotype_number;
		round=(read_number+32-1)/32;
		int round_size;
		for(int i=0;i<round;i++)
		{
			round_size=(read_number>32)?32: read_number;
			read_number=(read_number>32)?read_number-32:0; // read_num is the remaining length at this round
			char read_base;
			float M=1.0f; //now 
			float Qm,Qm_1,alpha,delta,xiksi;//thet;
			if(thread_in_warp<round_size ) // tid is from 0 ~ round_size-1
			{
				read_base=read_base_array[thread_in_warp+32*i];
				Qm=parameter_array[thread_in_warp+32*i];
				delta=parameter_array[thread_in_warp+32*i+skip];
				Qm_1=M-Qm;
				Qm=fdividef(Qm,3.0f);
				xiksi=parameter_array[thread_in_warp+32*i+2*skip];
				alpha=parameter_array[thread_in_warp+32*i+3*skip];
				//epsion=0.1;					
				//beta=0.9;
								//printf("%d %e %e %e %e %e %e \n",threadIdx.x, Qm_1, Qm, alpha, beta, delta, xiksi);
			}
			//why not use else break;?  Because we use __syncthreads() we need to make sure that all threads could reach that point
			M=0;
			float I=0; //now
			float D=0; //now
			float MMID=0;
			
			if(thread_in_warp==0&&i==0) MMID=__fmul_rn(0.9,D_0); // Just in the first round, it need to be D_0
			
			int current_haplotype_id=0;
			for(int j=0;j<round_size+haplotype_number-1;j++)
			{ 
				int aa=j-thread_in_warp;	
				float MM,DD,II;
				if( aa>=0 && (current_haplotype_id<haplotype_number))
				{
					
					if(thread_in_warp==0)
					{	
					if(i>0)
					{
					MM=MG[current_haplotype_id];
					II=IG[current_haplotype_id];
					DD=DG[current_haplotype_id];
					}
					else
					{
					MM=0;
					II=0;
					DD=D_0;
					}
					}	
					char haplotype_base_each=haplotype_base_in_char[warp_index_in_block*500+current_haplotype_id];
					float aa=(haplotype_base_each==read_base)? Qm_1:Qm;
				
					float MID=__fadd_rn(II,DD);
					float DDM=__fmul_rn(M,xiksi);
					float IIMI=__fmul_rn(II,0.1);
					M=__fmul_rn(aa,MMID);
					
					float MIIDD=__fmul_rn(0.9,MID);
					D=__fmaf_rn(D,0.1,DDM);
					I=__fmaf_rn(MM,delta,IIMI);
					current_haplotype_id++;
					II=I;
					DD=D;
					MMID=__fmaf_rn(alpha,MM,MIIDD);
					MM=M;
				 }
				if(thread_in_warp==round_size-1 && i<round-1) // tid is the last thread but there are more round
                                {
                                        MG[current_haplotype_id-1]=M;
                                        IG[current_haplotype_id-1]=I;
                                        DG[current_haplotype_id-1]=D;
                                }

				if(thread_in_warp==round_size-1 && i==round-1)
					result_block=__fadd_rn(result_block,__fadd_rn(M,I));
				MM=__shfl_up(MM,1);
                                II=__shfl_up(II,1);
                                DD=__shfl_up(DD,1);
			}
		}
		if(thread_in_warp==round_size-1) 
		{
			result[offset]=result_block;
		}	
		offset+=blockDim.x*gridDim.x/32;	
	 }
}

struct InputData
{
int read_size;
char read_base[260];
char base_quals[260];
char ins_quals[260];
char del_quals[260];
char gcp_quals[260];
int haplotype_size;
char haplotype_base[500];
};

	int main(int argc, char * argv[])
	{
		int INI=(log10f((std::numeric_limits<float>::max() / 16)));
		hipFree(0);	
		float * MG;
            	float * DG;
		float * IG;
		hipMalloc( (float **)& MG,sizeof(float) *4*150*500*3);
	        DG=MG+150*4*500;// ????
            	IG=DG+150*4*500;  //?????
			
		int size_each_for=4000000;
		//scanf("%d", &size_each_for);
		struct timespec start,finish,start_all,finish_all;
		double  computation_time=0,mem_cpy_time=0,read_time=0, data_prepare=0;
		FILE * file;
	//	file=fopen("a.txt","r");
	//	file=fopen("pairHMM_input_store.txt","r");
	//	file=fopen("32_data.txt","r");
		file=fopen("/data/04068/sren/dir_chromosome-10/b.txt", "r");
	//	file=fopen(argv[1],"r");
		//printf("OK\n");
		int size;
		fscanf(file,"%d",&size);
		clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
		float ph2pr_h[128];
		for(int i=0;i<128;i++)
		{
			ph2pr_h[i]=powf(10.f, -((float)i) / 10.f);
		}
		
		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);	
		data_prepare+=diff(start,finish);

		int total=0;
		float  read_read, haplotype_haplotype;
		double total_time=0;
		while(!feof(file))
		{
			total+=size;
			char useless;
			useless=fgetc(file);
			
			clock_gettime(CLOCK_MONOTONIC_RAW,&start); 
			InputData *inputdata=(InputData* )malloc(size*(sizeof(InputData)));		
		//	int size_each_for=1000;
			for(int i=0;i<size;i++)
			{
				int read_size;
				fscanf(file,"%d\n",&inputdata[i].read_size);
				fscanf(file,"%s ",inputdata[i].read_base);
				read_size=inputdata[i].read_size;
			//	if(read_size>200) 
			//	printf("read size is bigger than 200: size is %d \n", read_size);
				read_read=read_size;
				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i]. base_quals[j]=(char)aa;
				}

				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i].ins_quals[j]=(char)aa;
				}
				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				 fscanf(file,"%d ",&aa);
				 inputdata[i].del_quals[j]=(char)aa;
				}

				for(int j=0;j<read_size;j++)
				{
				 int  aa;
				if(j<read_size-1) fscanf(file,"%d ",&aa);
				else  fscanf(file,"%d \n",&aa);
				 inputdata[i].gcp_quals[j]=(char)aa;
				}

				fscanf(file,"%d\n",&inputdata[i].haplotype_size);
				fscanf(file, "%s\n",inputdata[i].haplotype_base);
				haplotype_haplotype=inputdata[i].haplotype_size;
			}
			clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
			read_time+=diff(start,finish);
			
			float * result_h=(float *) malloc(sizeof(float)*size); //on cpu
		

	
			clock_gettime(CLOCK_MONOTONIC_RAW,&start_all);
			
			char * data_h_total;
            		char * result_d_total;
			
			//printf("size=%d\n",size *700* sizeof (char)+size*200*4*sizeof(float)+size*sizeof(NUM_ADD) );
			int memory_malloc_size=(size*260+127)/128*128; //read_base
			memory_malloc_size+=(size*500+127)/128*128; // haplotyp_base;
			memory_malloc_size+=(size*260*4+127)/128*128;//parameter1;
			memory_malloc_size+=(size*260*4+127)/128*128;//parameter2;
			memory_malloc_size+=(size*260*4+127)/128*128;//parameter3;
			memory_malloc_size+=(size*260*4+127)/128*128;//parameter4;
			memory_malloc_size+=(size*4+127)/128*128;//result;
			memory_malloc_size+=(size*sizeof(NUM_ADD)+127)/128*128;//NUM_ADD;
			
			
			data_h_total=(char*)malloc(memory_malloc_size); //on cpu 
			hipError_t err;
			err=hipMalloc( (char **) &result_d_total, memory_malloc_size);
                        if(err!=hipSuccess)
    			printf( "Error %d: %s!\n", err, hipGetErrorString(err) );
			//printf("%p   %p  \n", result_d_total,result_d_total+memory_malloc_size);      
			char * data_d_total=result_d_total+(size*sizeof(float)+127)/128*128;  //on GPU

			//int num_streams=(size+size_each_for-1)/size_each_for;
			//cudaStream_t * streams=(cudaStream_t *) malloc(num_streams*sizeof(cudaStream_t));
		       //for(int aaa=0;aaa<num_streams;aaa++)
 	               //cudaStreamCreate(&streams[aaa]);
			
			//for(int aaa=0;aaa<num_streams;aaa++)
			//{
			//int size_in_each=size_each_for;
			//if(aaa==num_streams-1)
			//	size_in_each=size-aaa*size_each_for;
			
			//char * data_h=data_h_total+base*1500*sizeof(char)+base*sizeof(NUM_ADD);
			//char * data_h_begin=data_h; 
			char * data_h=data_h_total;  //cpu
			char * data_h_begin=data_h;  //cpu
			NUM_ADD *data_num_add=(NUM_ADD *) (data_h); //cpu
			
			data_h=data_h+(size*sizeof(NUM_ADD)+127)/128*128; // it is 64*x .thus we donot need to worry about alignment.
		
			int data_size=0;
			for(int i=0;i<size;i++)
			{
				int read_size=inputdata[i].read_size;
				int skip=(sizeof(float)*read_size+128-1)/128*128/sizeof(float);
				//float * parameter=(float *) malloc(skip*sizeof(float)*4);
				float parameter[1040];
				for(int j=0;j<read_size;j++)
				{
				    parameter[j]= ph2pr_h[inputdata[i].base_quals[j]&127 ];     //QM
				    parameter[j+skip]=ph2pr_h[inputdata[i].ins_quals[j]&127];      //Qi
				    parameter[j+skip*2]=ph2pr_h[inputdata[i].del_quals[j]&127];    //QD
				    parameter[j+skip*3]=1.0f-ph2pr_h[((int)(inputdata[i].ins_quals[j]&127)+(int)(inputdata[i].del_quals[j]&127))&127];  //alpha
			//	printf("%e %e %e %e\n", parameter[j],parameter[j+read_size], parameter[j+read_size*2],parameter[j+read_size*3]);
				}	
				
				char read_base_new[260];
				for(int j=0;j<read_size;j++)
				{	
				read_base_new[j]=inputdata[i].read_base[j];
				}	
		
				int haplotype_new_size=(inputdata[i].haplotype_size+4-1)/4;
				char4 haplotype_base_new[150];;
				for(int j=0;j<haplotype_new_size;j++)
				{
					haplotype_base_new[j].x=inputdata[i].haplotype_base[j*4];
					if(j*4+1<inputdata[i].haplotype_size)
					haplotype_base_new[j].y=inputdata[i].haplotype_base[j*4+1];
					if(j*4+2<inputdata[i].haplotype_size)
					haplotype_base_new[j].z=inputdata[i].haplotype_base[j*4+2];
					if(j*4+3<inputdata[i].haplotype_size)
					haplotype_base_new[j].w=inputdata[i].haplotype_base[j*4+3];			
				}

				data_num_add[i].read_haplotype_number.x=inputdata[i].read_size;
				data_num_add[i].read_haplotype_number.y=inputdata[i].haplotype_size;
				data_num_add[i].address_array=data_size;
					
				//read base
				memcpy(data_h,read_base_new,sizeof(char)*read_size);
				data_h+=(read_size+128-1)/128*128;
				data_size+=(read_size+128-1)/128*128;
				//printf("data_size=%d\n", data_size);
				//Parameter
				memcpy(data_h,parameter,sizeof(float) *skip*4);
				data_h+=sizeof(float) *skip*4;
				data_size+=sizeof(float) *skip*4;
				//printf("data_size=%d\n", data_size);
				
				//haplotype
				memcpy(data_h,haplotype_base_new,sizeof(char4)* haplotype_new_size);
				data_h+=(haplotype_new_size*sizeof(char4)+128-1)/128*128;
				data_size+=(haplotype_new_size*sizeof(char4)+128-1)/128*128;
				//printf("data_size=%d\n", data_size);
			}
			//printf("before call\n");
		
				
			int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+127)/128*128;			
			char * data_d;
			float * result_d=(float *) (result_d_total);	
		
			NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
			data_d=data_d_total+(sizeof(NUM_ADD)*size+127)/128*128;
                        clock_gettime(CLOCK_MONOTONIC_RAW,&start);
			err=hipMemcpy(data_d_total,data_h_begin,data_size_to_copy,hipMemcpyHostToDevice);
			if(err!=hipSuccess)
    			printf( "Error %d: %s!\n", err, hipGetErrorString(err) );

 			clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
			mem_cpy_time+=diff(start,finish);	
			//  call kernel
			int blocksize=128;
		//	int gridsize=1+(size/(blocksize/32));
			int gridsize=150;	
		
			clock_gettime(CLOCK_MONOTONIC_RAW,&start);
			pairHMM<<<gridsize,blocksize>>> (size,data_d,num_add_d,result_d,MG,DG,IG);
        		hipMemcpy(result_h,result_d_total,size*sizeof(float),hipMemcpyDeviceToHost);
 			
			clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
			computation_time+=diff(start,finish);
		// }
        //for(int aaa=0;aaa<num_streams;aaa++)
            //cudaStreamDestroy(streams[aaa]);
		    for(int i=0;i<size;i++)
 		float aa=(log10f((double)result_h[i]) - INI);  
	//	 printf("  i=%d  %e\n",i, result_h[i]);
	//	 printf("result_d_total=%p\n", result_d_total);
		
		free(data_h_total);
		err=hipFree(result_d_total);
		if(err!=hipSuccess)
    		printf( "Error %d: %s!\n", err, hipGetErrorString(err) );	
			
		clock_gettime(CLOCK_MONOTONIC_RAW,&finish_all);
		total_time+=diff(start_all, finish_all);		

		 free(inputdata);
		 free(result_h);
		 fscanf(file,"%d",&size);
		//printf("%d\n",size);
	//	if(total>10000000)
	//	break;
		}//end of while
		
		clock_gettime(CLOCK_MONOTONIC_RAW,&start);
    
		hipFree(MG);
	 	hipDeviceReset();
		clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
		mem_cpy_time+=diff(start,finish);//(finish1.tv_nsec-start1.tv_nsec)/1000000000.0;

	//	printf("size %d\n",total);
		printf("read_time=%e  initial_time=%e  computation_time= %e total_time=%e\n",read_time, data_prepare,computation_time, computation_time+mem_cpy_time);
		printf("total time=%e\n",total_time); 
	
	//   printf("GCUPS: %lf \n",  size*read_read*haplotype_haplotype/computation_time/1000000000);

		return 0;
	}

