#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <unistd.h>

#include "common.h"
#include "timer.h"

void nw_cpu(unsigned char* reference, unsigned char* query, int* matrix, unsigned int N) {
    for(int q = 0; q < N; ++q) {
        for (int r = 0; r < N; ++r) {
            // Get neighbors
            int top     = (q == 0)?((r + 1)*DELETION):(matrix[(q - 1)*N + r]);
            int left    = (r == 0)?((q + 1)*INSERTION):(matrix[q*N + (r - 1)]);
            int topleft = (q == 0)?(r*DELETION):((r == 0)?(q*INSERTION):(matrix[(q - 1)*N + (r - 1)]));
            // Find scores based on neighbors
            int insertion = top + INSERTION;
            int deletion  = left + DELETION;
            int match     = topleft + ((query[q] == reference[r])?MATCH:MISMATCH);
            // Select best score
            int max = (insertion > deletion)?insertion:deletion;
            max = (match > max)?match:max;
            matrix[q*N + r] = max;
        }
    }
}

void verify(int* matrix_cpu, int* matrix_gpu, unsigned int N) {
    for (unsigned int q = 0; q < N; ++q) {
        for (unsigned int r = 0; r < N; ++r) {
            if(matrix_cpu[q*N + r] != matrix_gpu[q*N + r]) {
                printf("\033[1;31mMismatch at q = %u, r = %u (CPU result = %d, GPU result = %d)\033[0m\n", q, r, matrix_cpu[q*N + r], matrix_gpu[q*N + r]);
                return;
            }
        }
    }
    printf("Verification succeeded\n");
}

void generateQuery(unsigned char* reference, unsigned char* query, unsigned int N) {
    const float PROB_MATCH = 0.80f;
    const float PROB_INS   = 0.10f;
    const float PROB_DEL   = 1.00f - PROB_MATCH - PROB_INS;
    assert(PROB_MATCH >= 0.00f && PROB_MATCH <= 1.00f);
    assert(PROB_INS   >= 0.00f && PROB_INS   <= 1.00f);
    assert(PROB_DEL   >= 0.00f && PROB_DEL   <= 1.00f);
    unsigned int r = 0, q = 0;
    while(r < N && q < N) {
        float prob = rand()*1.0f/RAND_MAX;
        if(prob < PROB_MATCH) {
            query[q++] = reference[r++]; // Match
        } else if(prob < PROB_MATCH + PROB_INS) {
            query[q++] = rand()%256; // Insertion
        } else {
            ++r; // Deletion
        }
    }
    while(q < N) {
        query[q++] = rand()%256; // Tail insertions
    }
}

int main(int argc, char**argv) {

    hipDeviceSynchronize();

    // Parse arguments
    unsigned int N = 32000;
    unsigned int runGPUVersion0 = 0;
    unsigned int runGPUVersion1 = 0;
    unsigned int runGPUVersion2 = 0;
    unsigned int runGPUVersion3 = 0;
    int opt;
    while((opt = getopt(argc, argv, "N:0123")) >= 0) {
        switch(opt) {
            case 'N': N = atoi(optarg);     break;
            case '0': runGPUVersion0 = 1;   break;
            case '1': runGPUVersion1 = 1;   break;
            case '2': runGPUVersion2 = 1;   break;
            case '3': runGPUVersion3 = 1;   break;
            default:  fprintf(stderr, "\nUnrecognized option!\n");
                      exit(0);
        }
    }

    // Allocate memory and initialize data
    Timer timer;
    unsigned char* reference = (unsigned char*) malloc(N*sizeof(unsigned char));
    unsigned char* query = (unsigned char*) malloc(N*sizeof(unsigned char));
    int* matrix_cpu = (int*) malloc(N*N*sizeof(int));
    int* matrix_gpu = (int*) malloc(N*N*sizeof(int));
    for(unsigned int r = 0; r < N; ++r) {
        reference[r] = rand()%256;
    }
    generateQuery(reference, query, N);
    
    //Open file for writing in appending mode
    FILE *fp_seq;
    fp_seq = fopen("runtimes_seq.txt", "a");

    // Compute on CPU
    startTime(&timer);
    nw_cpu(reference, query, matrix_cpu, N);
    stopTime(&timer);
    printElapsedTimeToFile(timer, fp_seq);
    printElapsedTime(timer, "CPU time", CYAN);

    if(runGPUVersion0 || runGPUVersion1 || runGPUVersion2 || runGPUVersion3) {

        // Allocate GPU memory
        startTime(&timer);
        unsigned char *reference_d;
        unsigned char *query_d;
        int *matrix_d;
        hipMalloc((void**) &reference_d, N*sizeof(unsigned char));
        hipMalloc((void**) &query_d, N*sizeof(unsigned char));
        hipMalloc((void**) &matrix_d, N*N*sizeof(int));
        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Allocation time");

        // Copy data to GPU
        startTime(&timer);
        hipMemcpy(reference_d, reference, N*sizeof(unsigned char), hipMemcpyHostToDevice);
        hipMemcpy(query_d, query, N*sizeof(unsigned char), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Copy to GPU time");

        if(runGPUVersion0) {

            // Reset
            hipMemset(matrix_d, 0, N*N*sizeof(int));
            hipDeviceSynchronize();
            
            // Open File
            FILE *fp_gpu0;
    		fp_gpu0 = fopen("runtimes_gpu0.txt", "a");

            // Compute on GPU with version 0
            startTime(&timer);
            nw_gpu0(reference_d, query_d, matrix_d, N);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTimeToFile(timer, fp_gpu0);
            printElapsedTime(timer, "GPU kernel time (version 0)", GREEN);

            // Copy data from GPU
            startTime(&timer);
            hipMemcpy(matrix_gpu, matrix_d, N*N*sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "Copy from GPU time");

            // Verify
            verify(matrix_cpu, matrix_gpu, N);
            
            // Close file
            fclose(fp_gpu0);

        }

        if(runGPUVersion1) {

            // Reset
            hipMemset(matrix_d, 0, N*N*sizeof(int));
            hipDeviceSynchronize();
            
            // Open File
            FILE *fp_gpu1;
    		fp_gpu1 = fopen("runtimes_gpu1.txt", "a");

            // Compute on GPU with version 1
            startTime(&timer);
            nw_gpu1(reference_d, query_d, matrix_d, N);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTimeToFile(timer, fp_gpu1);
            printElapsedTime(timer, "GPU kernel time (version 1)", GREEN);

            // Copy data from GPU
            startTime(&timer);
            hipMemcpy(matrix_gpu, matrix_d, N*N*sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "Copy from GPU time");

            // Verify
            verify(matrix_cpu, matrix_gpu, N);
            
            // Close file
            fclose(fp_gpu1);

        }

        if(runGPUVersion2) {

            // Reset
            hipMemset(matrix_d, 0, N*N*sizeof(int));
            hipDeviceSynchronize();
            
            // Open File
            FILE *fp_gpu2;
    		fp_gpu2 = fopen("runtimes_gpu2.txt", "a");

            // Compute on GPU with version 2
            startTime(&timer);
            nw_gpu2(reference_d, query_d, matrix_d, N);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTimeToFile(timer, fp_gpu2);
            printElapsedTime(timer, "GPU kernel time (version 2)", GREEN);

            // Copy data from GPU
            startTime(&timer);
            hipMemcpy(matrix_gpu, matrix_d, N*N*sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "Copy from GPU time");

            // Verify
            verify(matrix_cpu, matrix_gpu, N);
            
            // Close file
            fclose(fp_gpu2);

        }

        if(runGPUVersion3) {

            // Reset
            hipMemset(matrix_d, 0, N*N*sizeof(int));
            hipDeviceSynchronize();
            
            // Open File
            FILE *fp_gpu3;
    		fp_gpu3 = fopen("runtimes_gpu3.txt", "a");

            // Compute on GPU with version 3
            startTime(&timer);
            nw_gpu3(reference_d, query_d, matrix_d, N);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTimeToFile(timer, fp_gpu3);
            printElapsedTime(timer, "GPU kernel time (version 3)", GREEN);

            // Copy data from GPU
            startTime(&timer);
            hipMemcpy(matrix_gpu, matrix_d, N*N*sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            stopTime(&timer);
            printElapsedTime(timer, "Copy from GPU time");

            // Verify
            verify(matrix_cpu, matrix_gpu, N);
            
            // Close file
            fclose(fp_gpu3);

        }

        // Free GPU memory
        startTime(&timer);
        hipFree(reference_d);
        hipFree(query_d);
        hipFree(matrix_d);
        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Deallocation time");

    }
    
    //Close File
    fclose(fp_seq);

    // Free memory
    free(reference);
    free(query);
    free(matrix_cpu);
    free(matrix_gpu);

    return 0;

}

